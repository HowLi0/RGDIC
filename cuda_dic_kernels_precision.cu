#include "hip/hip_runtime.h"
#include "cuda_dic_kernel_precision.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <cmath>

using namespace cooperative_groups;

// 使用双精度常量确保与CPU版本完全一致
__constant__ double c_subsetRadius;
__constant__ int c_subsetSize;
__constant__ int c_imageWidth;
__constant__ int c_imageHeight;
__constant__ int c_numParams;
__constant__ double c_convergenceThreshold;
__constant__ int c_maxIterations;

// 双三次插值的三次样条核函数
__device__ __forceinline__ double cubicKernel(double t) {
    double abs_t = fabs(t);
    if (abs_t <= 1.0) {
        return 1.0 - 2.0 * abs_t * abs_t + abs_t * abs_t * abs_t;
    } else if (abs_t <= 2.0) {
        return 4.0 - 8.0 * abs_t + 5.0 * abs_t * abs_t - abs_t * abs_t * abs_t;
    } else {
        return 0.0;
    }
}

// 高精度双线性插值，完全匹配CPU版本
// 双线性插值公式： f(x,y) = f(x₁,y₁)·(1-fx)·(1-fy) + f(x₂,y₁)·fx·(1-fy) + f(x₁,y₂)·(1-fx)·fy + f(x₂,y₂)·fx·fy
/*
            (x1,y1) -------- (x2,y1)
            |                |
            |     (x,y)      |   
            |                |
            (x1,y2) -------- (x2,y2)
*/   
/*
               四个邻近像素的访问
image[y1 * width + x1]  // 左上角 (x1, y1)
image[y1 * width + x2]  // 右上角 (x2, y1) = (x1+1, y1)
image[y2 * width + x1]  // 左下角 (x1, y2) = (x1, y1+1)
image[y2 * width + x2]  // 右下角 (x2, y2) = (x1+1, y1+1)
*/
__device__ __forceinline__ double precisionBilinearInterpolation(double x, double y, const double* image, int width, int height) {
    // 边界检查，与CPU版本完全一致
    if (x < 0.0 || x >= width - 1.0 || y < 0.0 || y >= height - 1.0) {
        return 0.0;
    }
    
    // 获取整数和小数部分，使用与CPU相同的方法                          
    int x1 = static_cast<int>(x);
    int y1 = static_cast<int>(y);
    int x2 = x1 + 1;
    int y2 = y1 + 1;
    
    double fx = x - static_cast<double>(x1);// x 方向的权重
    double fy = y - static_cast<double>(y1);// y 方向的权重
    
    // 双线性插值，与CPU版本完全一致的公式
    double val = (1.0 - fx) * (1.0 - fy) * image[y1 * width + x1] +
                fx * (1.0 - fy) * image[y1 * width + x2] +
                (1.0 - fx) * fy * image[y2 * width + x1] +
                fx * fy * image[y2 * width + x2];
    
    return val;
}

// 高精度双三次插值，使用16点邻域
__device__ __forceinline__ double precisionBicubicInterpolation(double x, double y, const double* image, int width, int height) {
    // 边界检查，确保有足够的邻域进行双三次插值
    if (x < 1.0 || x >= width - 2.0 || y < 1.0 || y >= height - 2.0) {
        // 如果在边界附近，回退到双线性插值
        return precisionBilinearInterpolation(x, y, image, width, height);
    }
    
    // 获取整数和小数部分
    int x0 = static_cast<int>(floor(x));
    int y0 = static_cast<int>(floor(y));
    double fx = x - static_cast<double>(x0);
    double fy = y - static_cast<double>(y0);
    
    double result = 0.0;
    
    // 双三次插值使用4x4邻域
    for (int j = -1; j <= 2; j++) {
        for (int i = -1; i <= 2; i++) {
            int px = x0 + i;
            int py = y0 + j;
            
            // 边界检查
            if (px >= 0 && px < width && py >= 0 && py < height) {
                double weight_x = cubicKernel(fx - static_cast<double>(i));
                double weight_y = cubicKernel(fy - static_cast<double>(j));
                double weight = weight_x * weight_y;
                result += weight * image[py * width + px];
            }
        }
    }
    
    return result;
}

// 精确的点变形函数，与CPU版本完全一致
__device__ __forceinline__ void precisionWarpPoint(double x, double y, const double* warpParams, 
                                                   double& warpedX, double& warpedY, int numParams) {
    // 提取参数
    double u = warpParams[0];
    double v = warpParams[1];
    
    // 基础平移
    warpedX = x + u;
    warpedY = y + v;
    
    // 一阶形变参数（至少6个参数）
    if (numParams >= 6) {
        double dudx = warpParams[2];
        double dudy = warpParams[3];
        double dvdx = warpParams[4];
        double dvdy = warpParams[5];
        
        warpedX += dudx * x + dudy * y;
        warpedY += dvdx * x + dvdy * y;
    }
    
    // 二阶形变参数（12个参数）
    if (numParams >= 12) {
        double d2udx2 = warpParams[6];
        double d2udxdy = warpParams[7];
        double d2udy2 = warpParams[8];
        double d2vdx2 = warpParams[9];
        double d2vdxdy = warpParams[10];
        double d2vdy2 = warpParams[11];
        
        warpedX += 0.5 * d2udx2 * x * x + d2udxdy * x * y + 0.5 * d2udy2 * y * y;
        warpedY += 0.5 * d2vdx2 * x * x + d2vdxdy * x * y + 0.5 * d2vdy2 * y * y;
    }
}

// 精确的Sobel梯度计算，与CPU版本完全一致
__device__ __forceinline__ void computeSobelGradients(const double* image, int x, int y, int width, int height,
                                                      double& gradX, double& gradY) {
    gradX = 0.0;
    gradY = 0.0;
    
    // 边界检查
    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        // Sobel X核: [-1 0 1; -2 0 2; -1 0 1] / 8
        gradX = (-image[(y-1)*width + (x-1)] + image[(y-1)*width + (x+1)]
                -2.0*image[y*width + (x-1)] + 2.0*image[y*width + (x+1)]
                -image[(y+1)*width + (x-1)] + image[(y+1)*width + (x+1)]) / 8.0;
        
        // Sobel Y核: [-1 -2 -1; 0 0 0; 1 2 1] / 8  
        gradY = (-image[(y-1)*width + (x-1)] - 2.0*image[(y-1)*width + x] - image[(y-1)*width + (x+1)]
                +image[(y+1)*width + (x-1)] + 2.0*image[(y+1)*width + x] + image[(y+1)*width + (x+1)]) / 8.0;
    }
}

// 精确的ZNCC计算，与CPU版本完全一致
__device__ double computePrecisionZNCC(const double* refImage, const double* defImage,
                                      Point2D centerPoint, const double* warpParams,
                                      int imageWidth, int imageHeight, int subsetRadius, int numParams) {
    
    double sumRef = 0.0, sumDef = 0.0;
    double sumRefSq = 0.0, sumDefSq = 0.0;
    double sumRefDef = 0.0;
    int count = 0;
    
    // 遍历子集中的每个像素，与CPU版本完全一致
    for (int ly = -subsetRadius; ly <= subsetRadius; ly++) {
        for (int lx = -subsetRadius; lx <= subsetRadius; lx++) {
            // 参考图像中的像素位置
            int refX = centerPoint.x + lx;
            int refY = centerPoint.y + ly;
            
            // 边界检查
            if (refX >= 0 && refX < imageWidth && refY >= 0 && refY < imageHeight) {
                double refIntensity = refImage[refY * imageWidth + refX];
                
                // 计算变形后的点
                double warpedX, warpedY;
                precisionWarpPoint(static_cast<double>(lx), static_cast<double>(ly), 
                                 warpParams, warpedX, warpedY, numParams);
                
                // 变形图像中的像素位置
                double defImgX = static_cast<double>(centerPoint.x) + warpedX;
                double defImgY = static_cast<double>(centerPoint.y) + warpedY;
                
                // 边界检查
                if (defImgX >= 0.0 && defImgX < imageWidth - 1.0 && defImgY >= 0.0 && defImgY < imageHeight - 1.0) {
                    double defIntensity = precisionBilinearInterpolation(defImgX, defImgY, defImage, imageWidth, imageHeight);
                    
                    // 累积统计量
                    sumRef += refIntensity;
                    sumDef += defIntensity;
                    sumRefSq += refIntensity * refIntensity;
                    sumDefSq += defIntensity * defIntensity;
                    sumRefDef += refIntensity * defIntensity;
                    count++;
                }
            }
        }
    }
    
    // 计算ZNCC，与CPU版本完全一致
    if (count > 0) {
        double meanRef = sumRef / static_cast<double>(count);
        double meanDef = sumDef / static_cast<double>(count);
        double varRef = sumRefSq / static_cast<double>(count) - meanRef * meanRef;
        double varDef = sumDefSq / static_cast<double>(count) - meanDef * meanDef;
        double covar = sumRefDef / static_cast<double>(count) - meanRef * meanDef;
        
        // 防止除零，与CPU版本一致的阈值
        if (varRef > 1e-10 && varDef > 1e-10) {
            // 返回 1 - ZNCC 以转换为最小化问题
            return 1.0 - (covar / sqrt(varRef * varDef));
        }
    }
    
    return 1e10; // 错误情况，与CPU版本一致
}

// 高精度QR分解求解线性方程组 - 更稳定的数值方法
__device__ bool solvePrecisionLinearSystemQR(const double* A, const double* b, double* x, int n) {
    // 复制矩阵到局部内存
    double Q[144]; // 最大12x12矩阵 - 正交矩阵
    double R[144]; // 最大12x12矩阵 - 上三角矩阵
    double bb[12]; // 最大12维向量
    
    // 初始化
    for (int i = 0; i < n * n; i++) {
        Q[i] = A[i]; // 初始将A复制到Q
        R[i] = 0.0;
    }
    for (int i = 0; i < n; i++) {
        bb[i] = b[i];
        x[i] = 0.0;
    }
    
    // Modified Gram-Schmidt QR分解
    for (int j = 0; j < n; j++) {
        // 计算列向量的范数
        double norm = 0.0;
        for (int i = 0; i < n; i++) {
            norm += Q[i * n + j] * Q[i * n + j];
        }
        norm = sqrt(norm);
        
        // 检查数值稳定性
        if (norm < 1e-14) {
            return false; // 矩阵奇异
        }
        
        R[j * n + j] = norm;
        
        // 归一化列向量
        for (int i = 0; i < n; i++) {
            Q[i * n + j] /= norm;
        }
        
        // 计算与后续列的内积并正交化
        for (int k = j + 1; k < n; k++) {
            double dot = 0.0;
            for (int i = 0; i < n; i++) {
                dot += Q[i * n + j] * Q[i * n + k];
            }
            R[j * n + k] = dot;
            
            // 从后续列中减去投影
            for (int i = 0; i < n; i++) {
                Q[i * n + k] -= dot * Q[i * n + j];
            }
        }
    }
    
    // 计算 Q^T * b
    double QtB[12];
    for (int i = 0; i < n; i++) {
        QtB[i] = 0.0;
        for (int j = 0; j < n; j++) {
            QtB[i] += Q[j * n + i] * bb[j]; // Q^T[i][j] = Q[j][i]
        }
    }
    
    // 后向替换求解 R * x = Q^T * b
    for (int i = n - 1; i >= 0; i--) {
        double sum = 0.0;
        for (int j = i + 1; j < n; j++) {
            sum += R[i * n + j] * x[j];
        }
        
        // 检查对角元素避免除零
        if (fabs(R[i * n + i]) < 1e-14) {
            return false;
        }
        
        x[i] = (QtB[i] - sum) / R[i * n + i];
    }
    
    return true;
}

// 添加正则化的QR分解以处理病态矩阵
__device__ bool solvePrecisionLinearSystemRegularizedQR(const double* A, const double* b, double* x, int n) {
    // 复制矩阵并添加Tikhonov正则化
    double ARegularized[144];
    double regularization = 1e-8; // 正则化参数
    
    for (int i = 0; i < n * n; i++) {
        ARegularized[i] = A[i];
    }
    
    // 添加对角正则化项
    for (int i = 0; i < n; i++) {
        ARegularized[i * n + i] += regularization;
    }
    
    // 使用正则化矩阵进行QR分解
    return solvePrecisionLinearSystemQR(ARegularized, b, x, n);
}

// 主要的线性系统求解函数
__device__ bool solvePrecisionLinearSystem(const double* A, const double* b, double* x, int n) {
    // 首先尝试标准QR分解
    if (solvePrecisionLinearSystemQR(A, b, x, n)) {
        return true;
    }
    
    // 如果失败，尝试正则化QR分解
    return solvePrecisionLinearSystemRegularizedQR(A, b, x, n);
}

// 完全精确的ICGN优化核函数，与CPU版本算法完全一致
__global__ void precisionICGNOptimizationKernel(double* finalU, double* finalV, double* finalZNCC, bool* validMask,
                                               const double* refImage, const double* defImage,
                                               const Point2D* points, const double* initialParams,
                                               int numPoints, int imageWidth, int imageHeight,
                                               int subsetRadius, int numParams, int maxIterations,
                                               double convergenceThreshold) {
    
    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pointIdx >= numPoints) return;
    
    Point2D centerPoint = points[pointIdx];
    
    // 边界检查
    if (centerPoint.x < subsetRadius || centerPoint.x >= imageWidth - subsetRadius ||
        centerPoint.y < subsetRadius || centerPoint.y >= imageHeight - subsetRadius) {
        finalU[pointIdx] = 0.0;
        finalV[pointIdx] = 0.0;
        finalZNCC[pointIdx] = 1e10;
        validMask[pointIdx] = false;
        return;
    }
    
    // 使用局部内存存储参数和中间结果（减少内存使用）
    double warpParams[12] = {0};
    
    // 初始化参数
    for (int i = 0; i < numParams && i < 12; i++) {
        warpParams[i] = initialParams[pointIdx * numParams + i];
    }
    
    // Hessian矩阵存储
    double hessian[144] = {0}; // 最大12x12
    
    // 预计算Hessian矩阵，与CPU版本完全一致
    for (int i = 0; i < numParams; i++) {
        for (int j = i; j < numParams; j++) {
            double sum = 0.0;
            
            // 遍历子集计算Hessian元素
            for (int ly = -subsetRadius; ly <= subsetRadius; ly++) {
                for (int lx = -subsetRadius; lx <= subsetRadius; lx++) {
                    int refX = centerPoint.x + lx;
                    int refY = centerPoint.y + ly;
                    
                    if (refX >= 1 && refX < imageWidth - 1 && refY >= 1 && refY < imageHeight - 1) {
                        // 计算Sobel梯度
                        double gradX, gradY;
                        computeSobelGradients(refImage, refX, refY, imageWidth, imageHeight, gradX, gradY);
                        
                        // 计算shape function derivatives
                        double shapeFni = 0.0, shapeFnj = 0.0;
                        double x = static_cast<double>(lx);
                        double y = static_cast<double>(ly);
                        
                        // 计算第i个参数的shape function
                        if (i == 0) shapeFni = gradX; // du
                        else if (i == 1) shapeFni = gradY; // dv
                        else if (i == 2 && numParams >= 6) shapeFni = gradX * x; // du/dx
                        else if (i == 3 && numParams >= 6) shapeFni = gradX * y; // du/dy
                        else if (i == 4 && numParams >= 6) shapeFni = gradY * x; // dv/dx
                        else if (i == 5 && numParams >= 6) shapeFni = gradY * y; // dv/dy
                        else if (i == 6 && numParams >= 12) shapeFni = gradX * x * x * 0.5; // d²u/dx²
                        else if (i == 7 && numParams >= 12) shapeFni = gradX * x * y; // d²u/dxdy
                        else if (i == 8 && numParams >= 12) shapeFni = gradX * y * y * 0.5; // d²u/dy²
                        else if (i == 9 && numParams >= 12) shapeFni = gradY * x * x * 0.5; // d²v/dx²
                        else if (i == 10 && numParams >= 12) shapeFni = gradY * x * y; // d²v/dxdy
                        else if (i == 11 && numParams >= 12) shapeFni = gradY * y * y * 0.5; // d²v/dy²
                        
                        // 计算第j个参数的shape function
                        if (j == 0) shapeFnj = gradX; // du
                        else if (j == 1) shapeFnj = gradY; // dv
                        else if (j == 2 && numParams >= 6) shapeFnj = gradX * x; // du/dx
                        else if (j == 3 && numParams >= 6) shapeFnj = gradX * y; // du/dy
                        else if (j == 4 && numParams >= 6) shapeFnj = gradY * x; // dv/dx
                        else if (j == 5 && numParams >= 6) shapeFnj = gradY * y; // dv/dy
                        else if (j == 6 && numParams >= 12) shapeFnj = gradX * x * x * 0.5; // d²u/dx²
                        else if (j == 7 && numParams >= 12) shapeFnj = gradX * x * y; // d²u/dxdy
                        else if (j == 8 && numParams >= 12) shapeFnj = gradX * y * y * 0.5; // d²u/dy²
                        else if (j == 9 && numParams >= 12) shapeFnj = gradY * x * x * 0.5; // d²v/dx²
                        else if (j == 10 && numParams >= 12) shapeFnj = gradY * x * y; // d²v/dxdy
                        else if (j == 11 && numParams >= 12) shapeFnj = gradY * y * y * 0.5; // d²v/dy²
                        
                        sum += shapeFni * shapeFnj;
                    }
                }
            }
            
            hessian[i * numParams + j] = sum;
            hessian[j * numParams + i] = sum; // 对称矩阵
        }
    }
    
    // ICGN迭代优化，改进的收敛条件
    double prevZNCC = 1e10;
    bool converged = false;
    int iter = 0;
    
    for (iter = 0; iter < maxIterations && !converged; iter++) {
        // 计算当前ZNCC
        double currentZNCC = computePrecisionZNCC(refImage, defImage, centerPoint, warpParams, 
                                               imageWidth, imageHeight, subsetRadius, numParams);
        
        // 改进的收敛条件：ZNCC变化和参数变化都要考虑
        bool znccConverged = (iter > 0) && (fabs(currentZNCC - prevZNCC) < convergenceThreshold);
        
        // 如果ZNCC收敛，直接标记为成功
        if (znccConverged) {
            converged = true;
            finalZNCC[pointIdx] = currentZNCC;
            break;
        }
        
        prevZNCC = currentZNCC;
        
        // 计算误差向量，与CPU版本完全一致
        double errorVector[12] = {0};
        
        for (int ly = -subsetRadius; ly <= subsetRadius; ly++) {
            for (int lx = -subsetRadius; lx <= subsetRadius; lx++) {
                int refX = centerPoint.x + lx;
                int refY = centerPoint.y + ly;
                
                // 边界检查
                if (refX >= 0 && refX < imageWidth && refY >= 0 && refY < imageHeight) {
                    double refIntensity = refImage[refY * imageWidth + refX];
                    
                    // 变形点
                    double warpedX, warpedY;
                    precisionWarpPoint(static_cast<double>(lx), static_cast<double>(ly), 
                                     warpParams, warpedX, warpedY, numParams);
                    
                    double defImgX = static_cast<double>(centerPoint.x) + warpedX;
                    double defImgY = static_cast<double>(centerPoint.y) + warpedY;
                    
                    // 边界检查
                    if (defImgX >= 0.0 && defImgX < imageWidth - 1.0 && defImgY >= 0.0 && defImgY < imageHeight - 1.0) {
                        double defIntensity = precisionBilinearInterpolation(defImgX, defImgY, defImage, imageWidth, imageHeight);
                        double error = refIntensity - defIntensity;
                        
                        // 计算梯度和steepest descent 
                        if (refX >= 1 && refX < imageWidth - 1 && refY >= 1 && refY < imageHeight - 1) {
                            double gradX, gradY;
                            computeSobelGradients(refImage, refX, refY, imageWidth, imageHeight, gradX, gradY);
                            
                            double x = static_cast<double>(lx);
                            double y = static_cast<double>(ly);
                            
                            // 更新误差向量
                            if (numParams >= 2) {
                                errorVector[0] += error * gradX; // du
                                errorVector[1] += error * gradY; // dv
                            }
                            if (numParams >= 6) {
                                errorVector[2] += error * gradX * x; // du/dx
                                errorVector[3] += error * gradX * y; // du/dy
                                errorVector[4] += error * gradY * x; // dv/dx
                                errorVector[5] += error * gradY * y; // dv/dy
                            }
                            if (numParams >= 12) {
                                errorVector[6] += error * gradX * x * x * 0.5; // d²u/dx²
                                errorVector[7] += error * gradX * x * y; // d²u/dxdy
                                errorVector[8] += error * gradX * y * y * 0.5; // d²u/dy²
                                errorVector[9] += error * gradY * x * x * 0.5; // d²v/dx²
                                errorVector[10] += error * gradY * x * y; // d²v/dxdy
                                errorVector[11] += error * gradY * y * y * 0.5; // d²v/dy²
                            }
                        }
                    }
                }
            }
        }
        
        // 解线性方程组 H * deltaP = errorVector
        double deltaP[12] = {0};
        bool solved = solvePrecisionLinearSystem(hessian, errorVector, deltaP, numParams);
        
        if (!solved) {
            // 如果矩阵奇异，尝试使用当前参数作为结果
            // 而不是直接标记为失败
            finalZNCC[pointIdx] = currentZNCC;
            converged = (currentZNCC < 0.5); // 如果ZNCC还可以接受，就认为成功
            break;
        }
        
        // 更新参数
        for (int p = 0; p < numParams; p++) {
            warpParams[p] += deltaP[p];
        }
        
        // 检查参数更新的收敛性 - 更宽松的条件
        double deltaNorm = 0.0;
        for (int p = 0; p < numParams; p++) {
            deltaNorm += deltaP[p] * deltaP[p];
        }
        
        // 使用更宽松的参数收敛阈值
        if (sqrt(deltaNorm) < convergenceThreshold * 10.0) {
            converged = true;
            finalZNCC[pointIdx] = currentZNCC;
        }
    }
    
    // 改进的最终处理逻辑
    if (!converged) {
        if (iter >= maxIterations) {
            // 达到最大迭代次数，计算最终ZNCC
            double finalZncc = computePrecisionZNCC(refImage, defImage, centerPoint, warpParams, 
                                                   imageWidth, imageHeight, subsetRadius, numParams);
            finalZNCC[pointIdx] = finalZncc;
            
            // 如果ZNCC值可接受，仍然标记为有效
            converged = (finalZncc < 0.5); // 更宽松的有效性判断
        }
    }
    
    // 输出最终结果
    finalU[pointIdx] = warpParams[0];
    finalV[pointIdx] = warpParams[1];
    validMask[pointIdx] = converged;
    
    // 确保ZNCC有有效值
    if (!converged || finalZNCC[pointIdx] > 1e9) {
        finalZNCC[pointIdx] = 1e10;
    }
}

// 初始猜测核函数，与CPU版本完全一致
__global__ void precisionInitialGuessKernel(double* initialParams, double* initialZNCC, bool* validMask,
                                           const double* refImage, const double* defImage,
                                           const Point2D* points, int numPoints,
                                           int imageWidth, int imageHeight, int subsetRadius,
                                           int numParams, int searchRadius) {
    
    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pointIdx >= numPoints) return;
    
    Point2D centerPoint = points[pointIdx];
    
    // 初始化
    for (int i = 0; i < numParams; i++) {
        initialParams[pointIdx * numParams + i] = 0.0;
    }
    
    double bestZNCC = 1e10;
    int bestDx = 0, bestDy = 0;
    bool foundMatch = false;
    
    // 改进的网格搜索，使用更密集的搜索步长
    for (int dy = -searchRadius; dy <= searchRadius; dy += 1) { // 改为步长1，更精细
        for (int dx = -searchRadius; dx <= searchRadius; dx += 1) {
            Point2D testPoint = {centerPoint.x + dx, centerPoint.y + dy};
            
            // 边界检查
            if (testPoint.x >= subsetRadius && testPoint.x < imageWidth - subsetRadius &&
                testPoint.y >= subsetRadius && testPoint.y < imageHeight - subsetRadius) {
                
                // 创建简单的平移参数
                double testParams[12] = {0};
                testParams[0] = static_cast<double>(dx);
                testParams[1] = static_cast<double>(dy);
                
                // 计算ZNCC
                double testZNCC = computePrecisionZNCC(refImage, defImage, centerPoint, testParams, 
                                                     imageWidth, imageHeight, subsetRadius, numParams);
                
                // 更新最佳匹配
                if (testZNCC < bestZNCC) {
                    bestZNCC = testZNCC;
                    bestDx = dx;
                    bestDy = dy;
                    foundMatch = true;
                }
            }
        }
    }
    
    // 设置结果 - 更宽松的初始猜测有效性判断
    if (foundMatch && bestZNCC < 2.0) { // 更宽松的ZNCC阈值
        initialParams[pointIdx * numParams + 0] = static_cast<double>(bestDx);
        initialParams[pointIdx * numParams + 1] = static_cast<double>(bestDy);
        initialZNCC[pointIdx] = bestZNCC;
        validMask[pointIdx] = true;
    } else {
        // 即使没有找到好的匹配，也给一个零初始猜测
        initialParams[pointIdx * numParams + 0] = 0.0;
        initialParams[pointIdx * numParams + 1] = 0.0;
        initialZNCC[pointIdx] = bestZNCC;
        validMask[pointIdx] = true; // 仍然标记为有效，让后续优化尝试
    }
}

// 图像转换核函数 - 高精度版本
__global__ void precisionImageConvertKernel(double* dst, const unsigned char* src, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = static_cast<double>(src[idx]);
    }
}

// Host wrapper functions - 高精度版本
extern "C" {

void launchPrecisionICGNOptimizationKernel(double* finalU, double* finalV, double* finalZNCC, bool* validMask,
                                          const double* refImage, const double* defImage,
                                          const Point2D* points, const double* initialParams,
                                          int numPoints, int imageWidth, int imageHeight,
                                          int subsetRadius, int numParams, int maxIterations,
                                          double convergenceThreshold, hipStream_t stream) {
    
    // 计算网格和块大小
    int threadsPerBlock = 256; // 减少线程数以避免寄存器溢出
    int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
    
    // 启动高精度核函数
    precisionICGNOptimizationKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        finalU, finalV, finalZNCC, validMask,
        refImage, defImage, points, initialParams,
        numPoints, imageWidth, imageHeight,
        subsetRadius, numParams, maxIterations,
        convergenceThreshold
    );
    
    // 检查核函数启动是否成功
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Precision CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
}

void launchPrecisionInitialGuessKernel(double* initialParams, double* initialZNCC, bool* validMask,
                                      const double* refImage, const double* defImage,
                                      const Point2D* points, int numPoints,
                                      int imageWidth, int imageHeight, int subsetRadius,
                                      int numParams, int searchRadius, hipStream_t stream) {
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
    
    precisionInitialGuessKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        initialParams, initialZNCC, validMask,
        refImage, defImage, points, numPoints,
        imageWidth, imageHeight, subsetRadius,
        numParams, searchRadius
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Precision initial guess kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
}

void launchPrecisionImageConvertKernel(double* dst, const unsigned char* src, 
                                      int width, int height, hipStream_t stream) {
    int size = width * height;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    precisionImageConvertKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(dst, src, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Precision image convert kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
}

// CUDA kernel for high-precision displacement field interpolation with method selection
__global__ void precisionInterpolationKernel(double* interpU, double* interpV, unsigned char* interpMask,
                                            const double* sparseU, const double* sparseV, 
                                            const unsigned char* sparseMask, const unsigned char* roi,
                                            const Point2D* sparsePoints, int numSparsePoints,
                                            int width, int height, int step, int interpolationMethod) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPixels = width * height;
    
    if (idx >= totalPixels) return;
    
    int x = idx % width;
    int y = idx / width;
    
    // Only process points within ROI
    if (roi[y * width + x] == 0) return;
    
    if (interpolationMethod == 0) { // BILINEAR_INTERPOLATION - Simple bilinear for grid-based interpolation
        // For bilinear, we assume a regular grid structure and use 4-point interpolation
        // This is different from inverse distance weighting
        double resultU = 0.0, resultV = 0.0;
        bool hasValidResult = false;
        
        // Find 4 nearest grid points for bilinear interpolation
        double minDist = 1e10;
        int nearestIdx = -1;
        
        // First find the nearest point
        for (int i = 0; i < numSparsePoints; i++) {
            double dx = sparsePoints[i].x - x;
            double dy = sparsePoints[i].y - y;
            double dist = sqrt(dx*dx + dy*dy);
            if (dist < minDist) {
                minDist = dist;
                nearestIdx = i;
            }
        }
        
        if (nearestIdx >= 0 && minDist <= step * 1.5) {
            // Use nearest point value for simplicity in this bilinear implementation
            int sparseIdx = sparsePoints[nearestIdx].y * width + sparsePoints[nearestIdx].x;
            resultU = sparseU[sparseIdx];
            resultV = sparseV[sparseIdx];
            hasValidResult = true;
        }
        
        if (hasValidResult) {
            interpU[idx] = resultU;
            interpV[idx] = resultV;
            interpMask[idx] = 255;
        } else {
            interpU[idx] = 0.0;
            interpV[idx] = 0.0;
            interpMask[idx] = 0;
        }
        
    } else if (interpolationMethod == 2) { // INVERSE_DISTANCE_WEIGHTING - Original method
        // Use inverse distance weighting for interpolation (original algorithm)
        double weightSum = 0.0;
        double interpolatedU = 0.0;
        double interpolatedV = 0.0;
        
        double maxSearchRadius = step * 2.5; // Adaptive search radius
        bool foundNearbyPoints = false;
        
        // Search for nearby sparse points
        for (int i = 0; i < numSparsePoints; i++) {
            double dx = sparsePoints[i].x - x;
            double dy = sparsePoints[i].y - y;
            double dist = sqrt(dx*dx + dy*dy);
            
            if (dist <= maxSearchRadius) {
                foundNearbyPoints = true;
                
                // Handle exact matches (distance = 0)
                if (dist < 1e-6) {
                    int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                    interpolatedU = sparseU[sparseIdx];
                    interpolatedV = sparseV[sparseIdx];
                    weightSum = 1.0;
                    break;
                }
                
                double weight = 1.0 / (dist * dist); // Inverse distance squared
                int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                interpolatedU += weight * sparseU[sparseIdx];
                interpolatedV += weight * sparseV[sparseIdx];
                weightSum += weight;
            }
        }
        
        // Set interpolated values if we have enough nearby points
        if (foundNearbyPoints && weightSum > 0) {
            interpU[idx] = interpolatedU / weightSum;
            interpV[idx] = interpolatedV / weightSum;
            interpMask[idx] = 255;
        } else {
            interpU[idx] = 0.0;
            interpV[idx] = 0.0;
            interpMask[idx] = 0;
        }
        
    } else { // BICUBIC_INTERPOLATION - Surface fitting with bicubic interpolation
        // Check if we have enough boundary to perform bicubic interpolation
        if (x < 2 || x >= width - 2 || y < 2 || y >= height - 2) {
            // Fall back to bilinear method for boundary pixels
            double weightSum = 0.0;
            double interpolatedU = 0.0;
            double interpolatedV = 0.0;
            
            double maxSearchRadius = step * 2.5;
            bool foundNearbyPoints = false;
            
            for (int i = 0; i < numSparsePoints; i++) {
                double dx = sparsePoints[i].x - x;
                double dy = sparsePoints[i].y - y;
                double dist = sqrt(dx*dx + dy*dy);
                
                if (dist <= maxSearchRadius) {
                    foundNearbyPoints = true;
                    
                    if (dist < 1e-6) {
                        int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                        interpolatedU = sparseU[sparseIdx];
                        interpolatedV = sparseV[sparseIdx];
                        weightSum = 1.0;
                        break;
                    }
                    
                    double weight = 1.0 / (dist * dist);
                    int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                    interpolatedU += weight * sparseU[sparseIdx];
                    interpolatedV += weight * sparseV[sparseIdx];
                    weightSum += weight;
                }
            }
            
            if (foundNearbyPoints && weightSum > 0) {
                interpU[idx] = interpolatedU / weightSum;
                interpV[idx] = interpolatedV / weightSum;
                interpMask[idx] = 255;
            } else {
                interpU[idx] = 0.0;
                interpV[idx] = 0.0;
                interpMask[idx] = 0;
            }
        } else {
            // Use bicubic interpolation for interior points
            // Build local sparse displacement grid
            double localU[16], localV[16]; // 4x4 grid
            bool validLocal[16];
            int validCount = 0;
            
            // Initialize local arrays
            for (int i = 0; i < 16; i++) {
                localU[i] = 0.0;
                localV[i] = 0.0;
                validLocal[i] = false;
            }
            
            // Search for nearby sparse points to fill 4x4 grid around current point
            double searchRadius = step * 3.0; // Larger radius for bicubic
            for (int i = 0; i < numSparsePoints; i++) {
                double dx = sparsePoints[i].x - x;
                double dy = sparsePoints[i].y - y;
                double dist = sqrt(dx*dx + dy*dy);
                
                if (dist <= searchRadius) {
                    // Map to local 4x4 grid
                    int localX = static_cast<int>(round((dx + 1.5 * step) / step));
                    int localY = static_cast<int>(round((dy + 1.5 * step) / step));
                    
                    if (localX >= 0 && localX < 4 && localY >= 0 && localY < 4) {
                        int localIdx = localY * 4 + localX;
                        int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                        localU[localIdx] = sparseU[sparseIdx];
                        localV[localIdx] = sparseV[sparseIdx];
                        validLocal[localIdx] = true;
                        validCount++;
                    }
                }
            }
            
            // Need at least 9 points for reliable bicubic interpolation
            if (validCount >= 9) {
                // Perform bicubic interpolation on the local grid
                double fx = 1.5; // Relative position within the 4x4 grid
                double fy = 1.5;
                
                double resultU = 0.0, resultV = 0.0;
                double weightSum = 0.0;
                
                for (int j = 0; j < 4; j++) {
                    for (int i = 0; i < 4; i++) {
                        int localIdx = j * 4 + i;
                        if (validLocal[localIdx]) {
                            double weight_x = cubicKernel(fx - static_cast<double>(i));
                            double weight_y = cubicKernel(fy - static_cast<double>(j));
                            double weight = weight_x * weight_y;
                            
                            resultU += weight * localU[localIdx];
                            resultV += weight * localV[localIdx];
                            weightSum += weight;
                        }
                    }
                }
                
                if (weightSum > 0) {
                    interpU[idx] = resultU / weightSum;
                    interpV[idx] = resultV / weightSum;
                    interpMask[idx] = 255;
                } else {
                    interpU[idx] = 0.0;
                    interpV[idx] = 0.0;
                    interpMask[idx] = 0;
                }
            } else {
                // Fall back to inverse distance weighting if insufficient points
                double weightSum = 0.0;
                double interpolatedU = 0.0;
                double interpolatedV = 0.0;
                bool foundNearbyPoints = false;
                
                for (int i = 0; i < numSparsePoints; i++) {
                    double dx = sparsePoints[i].x - x;
                    double dy = sparsePoints[i].y - y;
                    double dist = sqrt(dx*dx + dy*dy);
                    
                    if (dist <= step * 2.5) {
                        foundNearbyPoints = true;
                        
                        if (dist < 1e-6) {
                            int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                            interpolatedU = sparseU[sparseIdx];
                            interpolatedV = sparseV[sparseIdx];
                            weightSum = 1.0;
                            break;
                        }
                        
                        double weight = 1.0 / (dist * dist);
                        int sparseIdx = sparsePoints[i].y * width + sparsePoints[i].x;
                        interpolatedU += weight * sparseU[sparseIdx];
                        interpolatedV += weight * sparseV[sparseIdx];
                        weightSum += weight;
                    }
                }
                
                if (foundNearbyPoints && weightSum > 0) {
                    interpU[idx] = interpolatedU / weightSum;
                    interpV[idx] = interpolatedV / weightSum;
                    interpMask[idx] = 255;
                } else {
                    interpU[idx] = 0.0;
                    interpV[idx] = 0.0;
                    interpMask[idx] = 0;
                }
            }
        }
    }
}

// CUDA kernel for high-precision strain field calculation using least squares
__global__ void precisionStrainCalculationKernel(double* strainExx, double* strainEyy, double* strainExy,
                                                unsigned char* strainMask, const double* u, const double* v,
                                                const unsigned char* validMask, int width, int height,
                                                int windowSize) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPixels = width * height;
    
    if (idx >= totalPixels) return;
    
    int x = idx % width;
    int y = idx / width;
    
    // Check boundaries
    if (x < windowSize || x >= width - windowSize || 
        y < windowSize || y >= height - windowSize) {
        strainExx[idx] = 0.0;
        strainEyy[idx] = 0.0;
        strainExy[idx] = 0.0;
        strainMask[idx] = 0;
        return;
    }
    
    // Check if center point is valid
    if (validMask[idx] == 0) {
        strainExx[idx] = 0.0;
        strainEyy[idx] = 0.0;
        strainExy[idx] = 0.0;
        strainMask[idx] = 0;
        return;
    }
    
    // Collect valid neighboring points within the strain window
    int validCount = 0;
    double sumX = 0.0, sumY = 0.0, sumU = 0.0, sumV = 0.0;
    double sumX2 = 0.0, sumY2 = 0.0, sumXY = 0.0;
    double sumXU = 0.0, sumYU = 0.0, sumXV = 0.0, sumYV = 0.0;
    
    for (int dy = -windowSize; dy <= windowSize; dy++) {
        for (int dx = -windowSize; dx <= windowSize; dx++) {
            int nx = x + dx;
            int ny = y + dy;
            int nIdx = ny * width + nx;
            
            if (validMask[nIdx] > 0) {
                validCount++;
                double relX = static_cast<double>(dx);
                double relY = static_cast<double>(dy);
                double uVal = u[nIdx];
                double vVal = v[nIdx];
                
                sumX += relX;
                sumY += relY;
                sumU += uVal;
                sumV += vVal;
                sumX2 += relX * relX;
                sumY2 += relY * relY;
                sumXY += relX * relY;
                sumXU += relX * uVal;
                sumYU += relY * uVal;
                sumXV += relX * vVal;
                sumYV += relY * vVal;
            }
        }
    }
    
    // Need at least 6 points for 2D least squares fitting
    if (validCount < 6) {
        strainExx[idx] = 0.0;
        strainEyy[idx] = 0.0;
        strainExy[idx] = 0.0;
        strainMask[idx] = 0;
        return;
    }
    
    // Solve least squares system using normal equations
    // For u = a0 + a1*x + a2*y, we need: [sumX2 sumXY; sumXY sumY2] * [a1; a2] = [sumXU; sumYU]
    double n = static_cast<double>(validCount);
    double det = n * sumX2 * sumY2 + 2.0 * sumX * sumY * sumXY - n * sumXY * sumXY - sumX * sumX * sumY2 - sumY * sumY * sumX2;
    
    if (fabs(det) < 1e-12) {
        strainExx[idx] = 0.0;
        strainEyy[idx] = 0.0;
        strainExy[idx] = 0.0;
        strainMask[idx] = 0;
        return;
    }
    
    // Calculate strain components (derivatives)
    // du/dx and dv/dx, dv/dy
    double detInv = 1.0 / det;
    
    // For u displacement: solve for du/dx (coefficient of x)
    double b1U = sumXU - (sumX * sumU) / n;
    double b2U = sumYU - (sumY * sumU) / n;
    double A11 = sumX2 - (sumX * sumX) / n;
    double A12 = sumXY - (sumX * sumY) / n;
    double A22 = sumY2 - (sumY * sumY) / n;
    
    double dudx = (A22 * b1U - A12 * b2U) / (A11 * A22 - A12 * A12);
    double dudy = (A11 * b2U - A12 * b1U) / (A11 * A22 - A12 * A12);
    
    // For v displacement: solve for dv/dx and dv/dy
    double b1V = sumXV - (sumX * sumV) / n;
    double b2V = sumYV - (sumY * sumV) / n;
    
    double dvdx = (A22 * b1V - A12 * b2V) / (A11 * A22 - A12 * A12);
    double dvdy = (A11 * b2V - A12 * b1V) / (A11 * A22 - A12 * A12);
    
    // Calculate strain components
    strainExx[idx] = dudx;                    // Normal strain in x
    strainEyy[idx] = dvdy;                    // Normal strain in y
    strainExy[idx] = 0.5 * (dudy + dvdx);    // Shear strain
    strainMask[idx] = 255;
}

void launchPrecisionInterpolationKernel(double* interpU, double* interpV, unsigned char* interpMask,
                                       const double* sparseU, const double* sparseV, 
                                       const unsigned char* sparseMask, const unsigned char* roi,
                                       const Point2D* sparsePoints, int numSparsePoints,
                                       int width, int height, int step, int interpolationMethod, 
                                       hipStream_t stream) {
    
    int totalPixels = width * height;
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;
    
    precisionInterpolationKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        interpU, interpV, interpMask, sparseU, sparseV, sparseMask, roi,
        sparsePoints, numSparsePoints, width, height, step, interpolationMethod);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Precision interpolation kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
}

void launchPrecisionStrainCalculationKernel(double* strainExx, double* strainEyy, double* strainExy,
                                           unsigned char* strainMask, const double* u, const double* v,
                                           const unsigned char* validMask, int width, int height,
                                           int windowSize, hipStream_t stream) {
    
    int totalPixels = width * height;
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;
    
    precisionStrainCalculationKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        strainExx, strainEyy, strainExy, strainMask, u, v, validMask, 
        width, height, windowSize);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Precision strain calculation kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
}

} // extern "C"
